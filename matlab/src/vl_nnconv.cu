/** @file vl_nnconv.cu
 ** @brief Convolution block
 ** @author Andrea Vedaldi
 ** @author Michael Figurnov
 **/

/*
 Copyright (C) 2014 Andrea Vedaldi and Max Jaderberg.
 All rights reserved.

 This file is part of the VLFeat library and is made available under
 the terms of the BSD license (see the COPYING file).
 */

#include "bits/mexutils.h"
#include "bits/nnhelper.h"
#include "bits/im2col.hpp"

#include <assert.h>
#include <algorithm>

#include <blas.h>
#ifdef ENABLE_GPU
#include "bits/gpu.hpp"
#include <hipblas.h>
#endif

/* option codes */
enum {
  opt_stride = 0,
  opt_pad,
  opt_conv_indices,
  opt_microbatch_size,
  opt_der_filters,
  opt_der_biases,
  opt_verbose,
  opt_no_der_data,
  opt_no_der_filters,
  opt_no_der_biases,
} ;

/* options */
vlmxOption  options [] = {
  {"Stride",           1,   opt_stride             },
  {"Pad",              1,   opt_pad                },
  {"ConvIndices",      1,   opt_conv_indices       },
  {"MicrobatchSize",   1,   opt_microbatch_size    },
  {"DerFilters",       1,   opt_der_filters        },
  {"DerBiases",        1,   opt_der_biases         },
  {"Verbose",          0,   opt_verbose            },
  {"NoDerData",        0,   opt_no_der_data        },
  {"NoDerFilters",     0,   opt_no_der_filters     },
  {"NoDerBiases",      0,   opt_no_der_biases      },
  {0,                  0,   0                      }
} ;

/* ---------------------------------------------------------------- */
/*                                                            Cache */
/* ---------------------------------------------------------------- */

#ifdef ENABLE_GPU
bool cublasInitialized = false ;
hipblasHandle_t thisCublasHandle ;
#endif

bool persistentDataInitialized = false ;
PackedData temp ;
PackedData derOutputMasked;
PackedData outputMasked;
PackedData allOnes ;

void atExit()
{
  if (persistentDataInitialized) {
    packed_data_deinit (&temp)  ;
    packed_data_deinit (&derOutputMasked)  ;
    packed_data_deinit (&outputMasked)  ;
    packed_data_deinit (&allOnes)  ;
    persistentDataInitialized = false ;
  }
#ifdef ENABLE_GPU
  if (cublasInitialized) {
    hipblasDestroy(thisCublasHandle) ;
    cublasInitialized = false ;
  }
#endif
}

/* ---------------------------------------------------------------- */
/*                                                  Dispatcher func */
/* ---------------------------------------------------------------- */

static void
sgemv_dispatch(bool gpuMode,
               char op,
               ptrdiff_t m, ptrdiff_t n,
               float alpha,
               float const * a, ptrdiff_t lda,
               float const * x, ptrdiff_t incx,
               float beta,
               float * y, ptrdiff_t incy)
{
  if (!gpuMode) {
    sgemv(&op,
          &m, &n, &alpha,
          (float*)a, &lda,
          (float*)x, &incx,
          &beta,
          y, &incy) ;
  } else {
#ifdef ENABLE_GPU
    hipblasSgemv(thisCublasHandle,
                (op == 't') ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                (int)m, (int)n,
                &alpha,
                a, lda,
                x, (int)incx,
                &beta,
                y, (int)incy) ;
#endif
  }
}

static void
sgemm_dispatch(bool gpuMode,
               char op1, char op2,
               ptrdiff_t m, ptrdiff_t n, ptrdiff_t k,
               float alpha,
               float const * a, ptrdiff_t lda,
               float const * b, ptrdiff_t ldb,
               float beta,
               float * c, ptrdiff_t ldc)
{
  if (!gpuMode) {
    sgemm(&op1, &op2,
          &m, &n, &k,
          &alpha,
          (float*)a, &lda,
          (float*)b, &ldb,
          &beta,
          c, &ldc) ;
  } else {
#ifdef ENABLE_GPU
    hipblasSgemm(thisCublasHandle,
                (op1 == 't') ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                (op2 == 't') ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                (int)m, (int)n, (int)k,
                &alpha,
                a, (int)lda,
                b, (int)ldb,
                &beta,
                c, (int)ldc);
#endif
  }
}

static void
im2col_dispatch(bool gpuMode,
                float* stacked,
                float const* data,
                size_t width,
                size_t height,
                size_t depth,
                size_t windowWidth,
                size_t windowHeight,
                size_t strideX,
                size_t strideY,
                size_t padLeft,
                size_t padRight,
                size_t padTop,
                size_t padBottom)
{
  if (!gpuMode) {
    im2col_cpu<float>(stacked,
                      data,
                      width,
                      height,
                      depth,
                      windowWidth,
                      windowHeight,
                      strideX,
                      strideY,
                      padLeft,
                      padRight,
                      padTop,
                      padBottom) ;
  } else {
#ifdef ENABLE_GPU
    im2col_gpu<float>(stacked,
                      data,
                      width,
                      height,
                      depth,
                      windowWidth,
                      windowHeight,
                      strideX,
                      strideY,
                      padLeft,
                      padRight,
                      padTop,
                      padBottom) ;
#endif
  }
}

static void
im2col_indexed_dispatch(bool gpuMode,
                        float* stacked,
                        float const* data,
                        int const* im2colIndices,
                        int im2colIndicesLength,
                        size_t width,
                        size_t height,
                        size_t depth,
                        size_t size,
                        size_t windowWidth,
                        size_t windowHeight)
{
  if (!gpuMode) {
    im2col_indexed_cpu<float>(stacked,
                              data,
                              im2colIndices,
                              im2colIndicesLength,
                              width,
                              height,
                              depth,
                              size,
                              windowWidth,
                              windowHeight);
  } else {
#ifdef ENABLE_GPU
    im2col_indexed_gpu<float>(stacked,
                              data,
                              im2colIndices,
                              im2colIndicesLength,
                              width,
                              height,
                              depth,
                              size,
                              windowWidth,
                              windowHeight);
#endif
  }
}

static void
col2im_dispatch(bool gpuMode,
                float* data,
                float const* stacked,
                size_t width,
                size_t height,
                size_t depth,
                size_t windowWidth,
                size_t windowHeight,
                size_t strideX,
                size_t strideY,
                size_t padLeft,
                size_t padRight,
                size_t padTop,
                size_t padBottom)
{
  if (!gpuMode) {
    col2im_cpu<float>(data,
                      stacked,
                      width,
                      height,
                      depth,
                      windowWidth,
                      windowHeight,
                      strideX,
                      strideY,
                      padLeft,
                      padRight,
                      padTop,
                      padBottom) ;
  } else {
#ifdef ENABLE_GPU
    col2im_gpu<float>(data,
                      stacked,
                      width,
                      height,
                      depth,
                      windowWidth,
                      windowHeight,
                      strideX,
                      strideY,
                      padLeft,
                      padRight,
                      padTop,
                      padBottom) ;
#endif
  }
}

static void
col2im_indexed_dispatch(bool gpuMode,
                        float* data,
                        float const* stacked,
                        int const* im2colIndices,
                        int im2colIndicesLength,
                        size_t width,
                        size_t height,
                        size_t depth,
                        size_t size,
                        size_t windowWidth,
                        size_t windowHeight)
{
  if (!gpuMode) {
    col2im_indexed_cpu(data,
                       stacked,
                       im2colIndices,
                       im2colIndicesLength,
                       width,
                       height,
                       depth,
                       size,
                       windowWidth,
                       windowHeight);
  } else {
#ifdef ENABLE_GPU
    col2im_indexed_gpu(data,
                       stacked,
                       im2colIndices,
                       im2colIndicesLength,
                       width,
                       height,
                       depth,
                       size,
                       windowWidth,
                       windowHeight) ;
#endif
  }
}

static void
transpose23_dispatch(bool gpuMode,
                     float* transposed,
                     float const* data,
                     size_t d1,
                     size_t d2,
                     size_t d3)
{
  if (!gpuMode) {
    transpose23_cpu(transposed, data, d1, d2, d3);
  } else {
#ifdef ENABLE_GPU
    transpose23_gpu(transposed, data, d1, d2, d3) ;
#endif
  }
}

static void
copy_dispatch(bool gpuMode,
              float * dest,
              float const * src,
              size_t numElements)
{
  if (!gpuMode) {
    memcpy(dest, src, numElements * sizeof(float)) ;
  } else {
#ifdef ENABLE_GPU
    hipMemcpy(dest, src, numElements * sizeof(float), hipMemcpyDeviceToDevice) ;
#endif
  }
}

/* ---------------------------------------------------------------- */
/*                                                       MEX driver */
/* ---------------------------------------------------------------- */

enum {
  IN_DATA = 0, IN_FILTERS, IN_BIASES, IN_DEROUTPUT, IN_END
} ;

enum {
  OUT_RESULT = 0, OUT_DERFILTERS, OUT_DERBIASES, OUT_END
} ;

void mexFunction(int nout, mxArray *out[],
                 int nin, mxArray const *in[])
{
  /* inputs */
  PackedData data ;
  PackedData filters ;
  PackedData biases ;
  PackedData derOutput ;
  PackedData convIndices ;
  PackedData derFiltersInit ;
  PackedData derBiasesInit ;

  /* outputs */
  PackedData output ;
  PackedData derData  ;
  PackedData derFilters ;
  PackedData derBiases ;

  PackedDataGeometry outputGeom ;
  PackedDataGeometry derDataGeom  ;
  PackedDataGeometry derFiltersGeom ;
  PackedDataGeometry derBiasesGeom ;
  PackedDataGeometry tempGeom ;
  PackedDataGeometry derOutputMaskedGeom ;
  PackedDataGeometry outputMaskedGeom ;
  PackedDataGeometry allOnesGeom ;

  int strideX = 1 ;
  int strideY = 1 ;
  int padLeft = 0 ;
  int padRight = 0 ;
  int padTop = 0 ;
  int padBottom = 0 ;
  int numGroups = 1 ;
  int microbatchSize = 1 ;

#if ENABLE_GPU
  hipblasStatus_t stat;
  bool gpuMode = false ;
#else
  bool const gpuMode = false ;
#endif
  bool backMode = false ;
  bool hasFilters = false ;
  bool hasBiases = false ;
  bool fullyConnectedMode = false ;
  bool is_1x1 = false ;
  bool computeDerData = true ;
  bool computeDerFilters = true ;
  bool computeDerBiases = true ;
  bool convIndicesMode = false;
  bool derFiltersInitialized = false ;
  bool derBiasesInitialized = false ;

  int verbosity = 0 ;
  int opt ;
  int next = IN_END ;
  mxArray const *optarg ;

  packed_data_init_empty(&data) ;
  packed_data_init_empty(&filters) ;
  packed_data_init_empty(&biases) ;
  packed_data_init_empty(&derOutput) ;
  packed_data_init_empty(&convIndices) ;
  packed_data_init_empty(&output) ;
  packed_data_init_empty(&derData) ;
  packed_data_init_empty(&derFilters) ;
  packed_data_init_empty(&derBiases) ;
  packed_data_init_empty(&derFiltersInit) ;
  packed_data_init_empty(&derBiasesInit) ;
  if (!persistentDataInitialized) {
    packed_data_init_empty(&temp) ;
    packed_data_init_empty(&outputMasked) ;
    packed_data_init_empty(&allOnes) ;
    persistentDataInitialized = true ;
  }

  /* -------------------------------------------------------------- */
  /*                                            Check the arguments */
  /* -------------------------------------------------------------- */

  mexAtExit(atExit) ;

  if (nin < 3) {
    mexErrMsgTxt("There are less than three arguments.") ;
  }

  if (nin > 3 && vlmxIsString(in[3],-1)) {
    next = 3 ;
    backMode = 0 ;
  } else {
    backMode = (nin >= 4) ;
  }

  while ((opt = vlmxNextOption (in, nin, options, &next, &optarg)) >= 0) {
    switch (opt) {
      case opt_verbose :
        ++ verbosity ;
        break ;

      case opt_stride :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          mexErrMsgTxt("STRIDE is not a plain matrix.") ;
        }
        switch (mxGetNumberOfElements(optarg)) {
          case 1:
            strideY = (int)mxGetPr(optarg)[0] ;
            strideX = strideY ;
            break ;
          case 2:
            strideY = (int)mxGetPr(optarg)[0] ;
            strideX = (int)mxGetPr(optarg)[1] ;
            break ;
          default:
            mexErrMsgTxt("STRIDE has neither one nor two elements.") ;
        }
        break ;

      case opt_pad :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          mexErrMsgTxt("PAD is not a plain matrix.") ;
        }
        switch (mxGetNumberOfElements(optarg)) {
          case 1:
            padLeft = (int)mxGetPr(optarg)[0] ;
            padRight = padLeft ;
            padTop = padLeft ;
            padBottom = padLeft ;
            break ;
          case 4:
            padTop = (int)mxGetPr(optarg)[0] ;
            padBottom = (int)mxGetPr(optarg)[1] ;
            padLeft = (int)mxGetPr(optarg)[2] ;
            padRight = (int)mxGetPr(optarg)[3] ;
            break ;
          default:
            mexErrMsgTxt("STRIDE has neither one nor two elements.") ;
        }
        break ;

      case opt_conv_indices :
        if (mxGetNumberOfElements(optarg) != 0) {
          convIndicesMode = true;
          packed_data_init_with_array_int(&convIndices, optarg);
        }
        break;

      case opt_microbatch_size :
        if (mxGetNumberOfElements(optarg) == 1) {
          microbatchSize = (int)mxGetPr(optarg)[0] ;
        }
        break;

      case opt_der_filters :
        if (mxGetNumberOfElements(optarg) != 0) {
          derFiltersInitialized = true;
          packed_data_init_with_array(&derFiltersInit, optarg);
        }
        break;

      case opt_der_biases :
        if (mxGetNumberOfElements(optarg) != 0) {
          derBiasesInitialized = true;
          packed_data_init_with_array(&derBiasesInit, optarg);
        }
        break;

      case opt_no_der_data :
        computeDerData = VL_FALSE ;
        break ;

      case opt_no_der_filters :
        computeDerFilters = VL_FALSE ;
        break ;

      case opt_no_der_biases :
        computeDerBiases = VL_FALSE ;
        break ;

      default: break ;
    }
  }

  packed_data_init_with_array(&data, in[IN_DATA]) ;
  packed_data_init_with_array(&filters, in[IN_FILTERS]) ;
  packed_data_init_with_array(&biases, in[IN_BIASES]) ;
  if (backMode) { packed_data_init_with_array(&derOutput, in[IN_DEROUTPUT]) ; }

#if ENABLE_GPU
  gpuMode = (data.mode == matlabGpuArrayWrapper) ;
  if (gpuMode) {
    mxInitGPU() ;
    if (!cublasInitialized) {
      stat = hipblasCreate(&thisCublasHandle) ;
      if (stat != HIPBLAS_STATUS_SUCCESS) {
        mexErrMsgTxt("Could not initialize cuBLAS.") ;
      }
      cublasInitialized = true ;
    }
  }
#endif

  hasFilters = filters.geom.numElements > 0 ;
  hasBiases = biases.geom.numElements > 0 ;

  /* check for GPU/data class consistency */
  if (! hasFilters) {
    mexErrMsgTxt("FILTERS is empty.") ;
  }
  if (! packed_data_are_compatible(&data, &filters)) {
    mexErrMsgTxt("DATA and FILTERS are not both CPU or GPU arrays.") ;
  }
  if (hasBiases && ! packed_data_are_compatible(&data, &biases)) {
    mexErrMsgTxt("DATA and BIASES are not both CPU or GPU arrays.") ;
  }
  if (backMode && ! packed_data_are_compatible(&data, &derOutput)) {
    mexErrMsgTxt("DATA and DEROUTPUT are not both CPU or GPU arrays.") ;
  }
  if (data.geom.classID != mxSINGLE_CLASS) {
    mexErrMsgTxt("DATA is not of class SINGLE.");
  }
  if (filters.geom.classID != mxSINGLE_CLASS) {
    mexErrMsgTxt("FILTERS is not of class SINGLE.");
  }
  if (hasBiases && (biases.geom.classID != mxSINGLE_CLASS)) {
    mexErrMsgTxt("BIASES is not of class SINGLE.");
  }
  if (backMode && (derOutput.geom.classID != mxSINGLE_CLASS)) {
    mexErrMsgTxt("DEROUTPUT is not of class SINGLE.");
  }

  if (strideX < 1 || strideY < 1) {
    mexErrMsgTxt("At least one element of STRIDE is smaller than one.") ;
  }
  if (convIndicesMode && ! packed_data_are_compatible(&data, &convIndices)) {
    mexErrMsgTxt("DATA and CONVINDICES are not both CPU or GPU arrays.") ;
  }
  if (convIndicesMode && (convIndices.geom.classID != mxINT32_CLASS)) {
    mexErrMsgTxt("CONVINDICES is not of class INT32.");
  }

  if (convIndicesMode) {
    packed_data_geom_init(&outputGeom,
                          mxSINGLE_CLASS,
                          convIndices.geom.height,
                          convIndices.geom.width,
                          filters.geom.size,
                          data.geom.size) ;
  } else {
    packed_data_geom_init(&outputGeom,
                          mxSINGLE_CLASS,
                          (data.geom.height + (padTop+padBottom) - filters.geom.height)/strideY + 1,
                          (data.geom.width + (padLeft+padRight) - filters.geom.width)/strideX + 1,
                          filters.geom.size,
                          data.geom.size) ;
  }

  /* grouped filters */
  numGroups = data.geom.depth / filters.geom.depth ;

  /* if the output is 1x1 pixels, then there is no need to actually
   call im2col as it does not do anything
   */
  fullyConnectedMode = (!convIndicesMode &&
                        outputGeom.height == 1 &&
                        outputGeom.width == 1 &&
                        padTop == 0 &&
                        padBottom == 0 &&
                        padLeft == 0 &&
                        padRight == 0 &&
                        numGroups == 1) ;
  is_1x1 = (!convIndicesMode &&
            filters.geom.height == 1 &&
            filters.geom.width == 1 &&
            strideY == 1 &&
            strideX == 1 &&
            padTop == 0 &&
            padBottom == 0 &&
            padLeft == 0 &&
            padRight == 0);

  if (convIndicesMode) {
    if (convIndices.geom.depth != filters.geom.height*filters.geom.width) {
      mexErrMsgTxt("CONVINDICES depth is not compatible with filters.");
    }

    if (convIndices.geom.size != 1 && convIndices.geom.size != data.geom.size) {
      mexErrMsgTxt("CONVINDICES size should be equal either one, or the number of input images.");
    }
  }

  if (!is_1x1) {
    packed_data_geom_init
    (&tempGeom, mxSINGLE_CLASS,
     outputGeom.height,
     outputGeom.width,
     filters.geom.height*filters.geom.width*filters.geom.depth*numGroups,
     microbatchSize) ;
  } else {
    packed_data_geom_init (&tempGeom, mxSINGLE_CLASS,
                           0, 0, 0, 0) ;
  }

  if (convIndicesMode) {
    packed_data_geom_init
    (&outputMaskedGeom, mxSINGLE_CLASS,
     outputGeom.height,
     outputGeom.width,
     filters.geom.size,
     microbatchSize) ;
  } else {
    packed_data_geom_init (&outputMaskedGeom, mxSINGLE_CLASS,
                           0, 0, 0, 0) ;
  }

  if (false) {
    packed_data_geom_init (&derOutputMaskedGeom, mxSINGLE_CLASS,
                           outputGeom.height,
                           outputGeom.width,
                           filters.geom.size,
                           microbatchSize) ;
  } else {
    packed_data_geom_init (&derOutputMaskedGeom, mxSINGLE_CLASS,
                           0, 0, 0, 0) ;
  }

  derDataGeom = data.geom ;
  derFiltersGeom = filters.geom ;
  if (hasBiases) {
    if (fullyConnectedMode) {
      packed_data_geom_init (&allOnesGeom, mxSINGLE_CLASS,
                             1, 1,
                             1, data.geom.size) ;
    } else {
      packed_data_geom_init (&allOnesGeom, mxSINGLE_CLASS,
                             outputGeom.height,
                             outputGeom.width,
                             1, microbatchSize) ;
    }
    derBiasesGeom = biases.geom ;
  } else {
    packed_data_geom_init (&allOnesGeom, mxSINGLE_CLASS,
                           0, 0, 0, 0) ;
  }

  if (verbosity > 0) {
    mexPrintf("vl_nnconv: mode %s; %s\n", gpuMode?"gpu":"cpu", backMode?"backward":"forward") ;
    mexPrintf("vl_nnconv: stride: [%d %d], pad: [%d %d %d %d], numGroups: %d, has bias: %d, fully connected: %d, 1x1: %d, conv indices: %d, microbatchSize: %d\n",
              strideY, strideX,
              padTop, padBottom, padLeft, padRight,
              numGroups, hasBiases, fullyConnectedMode, is_1x1, convIndicesMode,
              microbatchSize) ;
    packed_data_geom_display(&data.geom, "vl_nnconv: data") ;
    packed_data_geom_display(&filters.geom, "vl_nnconv: filters") ;
    if (hasBiases) { packed_data_geom_display(&biases.geom, "vl_nnconv: biases") ; }
    if (backMode) {
      packed_data_geom_display(&derOutput.geom, "vl_nnconv: derOutput") ;
      packed_data_geom_display(&derOutputMaskedGeom, "vl_nnconv: derOutputMasked") ;
      packed_data_geom_display(&derOutputMasked.geom, "vl_nnconv: derOutputMasked (cached)") ;
      packed_data_geom_display(&derDataGeom, "vl_nnconv: derData") ;
      packed_data_geom_display(&derFiltersGeom, "vl_nnconv: derFilters") ;
      if (hasBiases) { packed_data_geom_display(&derBiasesGeom, "vl_nnconv: derBiases") ; }
    } else {
      packed_data_geom_display(&outputGeom, "vl_nnconv: output") ;
    }
    packed_data_geom_display(&tempGeom, "vl_nnconv: temp") ;
    packed_data_geom_display(&temp.geom, "vl_nnconv: temp (cached)") ;
    packed_data_geom_display(&outputMaskedGeom, "vl_nnconv: outputMasked") ;
    packed_data_geom_display(&outputMasked.geom, "vl_nnconv: outputMasked (cached)") ;
    packed_data_geom_display(&allOnesGeom, "vl_nnconv: allOnes") ;
    packed_data_geom_display(&allOnes.geom, "vl_nnconv: allOnes (cached)") ;
    if (convIndicesMode) {
      packed_data_geom_display(&convIndices.geom, "vl_nnconv: convIndices") ;
    }
  }

  if (backMode) {
    if (derOutput.geom.height != outputGeom.height ||
        derOutput.geom.width != outputGeom.width ||
        derOutput.geom.depth != filters.geom.size ||
        derOutput.geom.size != data.geom.size)
    {
      mexErrMsgTxt("DEROUTPUT dimensions are incompatible with X and FILTERS.") ;
    }
  }

  if (numGroups * filters.geom.depth != data.geom.depth) {
    mexErrMsgTxt("The filter depth does not divide the image depth.") ;
  }

  if (filters.geom.size % numGroups != 0) {
    mexErrMsgTxt("The number of filter groups does not divide the total number of filters.") ;
  }

  if (padLeft < 0 ||
      padRight < 0 ||
      padTop < 0 ||
      padBottom < 0) {
    mexErrMsgTxt("An element of PAD is negative.") ;
  }

  if (outputGeom.height == 0 || outputGeom.width == 0) {
    mexErrMsgTxt("FILTERS are larger than the DATA (including padding).") ;
  }

  if (filters.geom.height == 0 || filters.geom.width == 0 || filters.geom.depth == 0) {
    mexErrMsgTxt("A dimension of FILTERS is void.") ;
  }

  if (hasBiases) {
    if (biases.geom.numElements != filters.geom.size) {
      mexErrMsgTxt("The number of elements of BIASES is not the same as the number of filters.") ;
    }
  }

  /* -------------------------------------------------------------- */
  /*                                                    Do the work */
  /* -------------------------------------------------------------- */

  /* auxiliary buffers */
  if (hasBiases) {
    if (allOnes.memorySize < allOnesGeom.numElements * sizeof(float) ||
        (allOnes.mode == matlabGpuArray || allOnes.mode == matlabGpuArrayWrapper) != gpuMode) {
      packed_data_deinit (&allOnes) ;
      packed_data_init_with_geom (&allOnes, gpuMode, allOnesGeom, true, true, 1.0f) ;
    }
  }
  if (!fullyConnectedMode) {
    if (temp.memorySize < tempGeom.numElements * sizeof(float) ||
        (temp.mode == matlabGpuArray || temp.mode == matlabGpuArrayWrapper) != gpuMode) {
      packed_data_deinit (&temp) ;
      packed_data_init_with_geom (&temp, gpuMode, tempGeom, true, false, 0);
    }
  }
  if (derOutputMasked.memorySize < derOutputMaskedGeom.numElements * sizeof(float) ||
      (derOutputMasked.mode == matlabGpuArray || derOutputMasked.mode == matlabGpuArrayWrapper) != gpuMode) {
    packed_data_deinit (&derOutputMasked) ;
    packed_data_init_with_geom (&derOutputMasked, gpuMode, derOutputMaskedGeom, true, false, 0);
  }
  if (outputMasked.memorySize < outputMaskedGeom.numElements * sizeof(float) ||
      (outputMasked.mode == matlabGpuArray || outputMasked.mode == matlabGpuArrayWrapper) != gpuMode) {
    packed_data_deinit (&outputMasked) ;
    packed_data_init_with_geom (&outputMasked, gpuMode, outputMaskedGeom, true, false, 0);
  }
  if (!backMode) {
    packed_data_init_with_geom(&output, gpuMode, outputGeom, false, false, 0) ;
  } else {
    if (computeDerData) {
      packed_data_init_with_geom(&derData, gpuMode, derDataGeom, false, false, 0) ;
    }
    if (computeDerFilters) {
      packed_data_init_with_geom(&derFilters, gpuMode, derFiltersGeom, false, false, 0) ;
      if (derFiltersInitialized) {
        copy_dispatch(gpuMode, derFilters.memory, derFiltersInit.memory, derFilters.geom.numElements);;
      }
    }
    if (computeDerBiases && hasBiases) {
      packed_data_init_with_geom(&derBiases, gpuMode, derBiasesGeom, false, false, 0) ;
      if (derFiltersInitialized) {
        copy_dispatch(gpuMode, derBiases.memory, derBiasesInit.memory, derBiases.geom.numElements);;
      }
    }
  }

  if (fullyConnectedMode) {
    float alpha = 1 ;
    float beta = 0 ;
    ptrdiff_t filtersVolume = filters.geom.height*filters.geom.width*filters.geom.depth ;
    /* note: fullyConnectedMode also guarantees no padding, num filter groups = 1 */

    /* optimise fully-connected mode case */
    if (!backMode) {
      if (data.geom.size == 1) {
        /* one image in the stack */
        sgemv_dispatch(gpuMode, 't',
                       filtersVolume, filters.geom.size,
                       alpha,
                       filters.memory, filtersVolume,
                       data.memory, 1,
                       beta,
                       output.memory, 1) ;
      } else {
        /* multiple images in the stack */
        sgemm_dispatch(gpuMode, 't', 'n',
                       filters.geom.size, data.geom.size, filtersVolume,
                       alpha,
                       filters.memory, filtersVolume,
                       data.memory, filtersVolume,
                       beta,
                       output.memory, filters.geom.size) ;
      }
      if (hasBiases) {
        float beta = 1 ;
        ptrdiff_t q = 1 ;
        sgemm_dispatch(gpuMode, 'n', 'n',
                       filters.geom.size, data.geom.size, q,
                       alpha,
                       biases.memory, filters.geom.size,
                       allOnes.memory, q,
                       beta,
                       output.memory, filters.geom.size) ;
      }
    } else {
      /* back mode */
      if (computeDerFilters) {
        sgemm_dispatch(gpuMode, 'n', 't',
                       filtersVolume, filters.geom.size, data.geom.size,
                       alpha,
                       data.memory, filtersVolume,
                       derOutput.memory, filters.geom.size,
                       (float)(derFiltersInitialized > 0),
                       derFilters.memory, filtersVolume) ;
      }
      if (computeDerBiases && hasBiases) {
        ptrdiff_t q = 1 ;
        sgemm_dispatch(gpuMode, 'n', 't',
                       q, filters.geom.size, data.geom.size,
                       alpha,
                       allOnes.memory, q,
                       derOutput.memory, filters.geom.size,
                       (float)(derBiasesInitialized > 0),
                       derBiases.memory, q) ;
      }
      if (computeDerData) {
        sgemm_dispatch(gpuMode, 'n', 'n',
                       filtersVolume, data.geom.size, filters.geom.size,
                       alpha,
                       filters.memory, filtersVolume,
                       derOutput.memory, filters.geom.size,
                       beta,
                       derData.memory, filtersVolume) ;
      }
    }
  } else if (convIndicesMode) {
    // microbatchSize specifies the number of images to stack for GEMM
    const int numMicrobatches = (data.geom.size + microbatchSize - 1) / microbatchSize;
    for (int microbatchIdx = 0; microbatchIdx < numMicrobatches; ++microbatchIdx) {
      int image = microbatchIdx * microbatchSize;
      int numImages = (microbatchIdx != numMicrobatches - 1) ? microbatchSize : (data.geom.size - image);

      ptrdiff_t dataOffset = (data.geom.height*data.geom.width*data.geom.depth) * image ;
      ptrdiff_t outputOffset = (output.geom.height*output.geom.width*output.geom.depth) * image ;
      ptrdiff_t derDataOffset = (derData.geom.height*derData.geom.width*derData.geom.depth) * image ;
      ptrdiff_t derOutputOffset = (derOutput.geom.height*derOutput.geom.width*derOutput.geom.depth) * image ;
      ptrdiff_t m = outputGeom.height * outputGeom.width ; /* num output pixels */
      ptrdiff_t numRows = m * numImages ;
      ptrdiff_t n = filters.geom.size/numGroups ; /* num filters per group */
      ptrdiff_t k = filters.geom.height*filters.geom.width*filters.geom.depth ; /* filter volume */

      if (backMode) {
        if (numImages > 1) {
          transpose23_dispatch(gpuMode,
                               outputMasked.memory,
                               derOutput.memory + derOutputOffset,
                               m, derOutput.geom.depth, numImages) ;
        }

        float *curDerOutputMemory = numImages > 1 ? outputMasked.memory : derOutput.memory + derOutputOffset;

        /* compute derFilters dz/dF */
        if (computeDerFilters) {
          im2col_indexed_dispatch(gpuMode,
                                  temp.memory,
                                  data.memory + dataOffset,
                                  convIndices.memoryInt,
                                  convIndices.geom.numElements,
                                  data.geom.height, data.geom.width, data.geom.depth, numImages,
                                  filters.geom.height, filters.geom.width) ;
          for (int g = 0 ; g < numGroups ; ++ g) {
            ptrdiff_t filterGrpOffset = k * n * g ;
            ptrdiff_t tempGrpOffset = numRows * k * g ;
            ptrdiff_t derOutputGrpOffset = numRows * n * g ;
            float alpha = 1 ;
            float beta = (image > 0 || derFiltersInitialized) ; /* this saves init. the output array with 0 */
            sgemm_dispatch(gpuMode, 't', 'n',
                           k, n, numRows,
                           alpha,
                           temp.memory + tempGrpOffset, numRows,
                           curDerOutputMemory + derOutputGrpOffset, numRows,
                           beta,
                           derFilters.memory + filterGrpOffset, k) ;
          }
        }

        /* compute derData dz/dbias */
        if (computeDerBiases & hasBiases) {
          sgemv_dispatch(gpuMode, 't',
                         numRows, filters.geom.size,
                         1, /* alpha */
                         curDerOutputMemory, numRows,
                         allOnes.memory, 1,
                         (float)(image > 0 || derBiasesInitialized), /* beta */
                         derBiases.memory, 1) ;
        }

        /* compute derData dz/dx */
        if (computeDerData) {
          for (int g = 0 ; g < numGroups ; ++ g) {
            ptrdiff_t filterGrpOffset = k * n * g ;
            ptrdiff_t tempGrpOffset = numRows * k * g ;
            ptrdiff_t derOutputGrpOffset = numRows * n * g ;
            float alpha = 1 ;
            float beta = 0 ;
            sgemm_dispatch(gpuMode, 'n', 't',
                           numRows, k, n,
                           alpha,
                           curDerOutputMemory + derOutputGrpOffset, numRows,
                           filters.memory + filterGrpOffset, k,
                           beta,
                           temp.memory + tempGrpOffset,
                           numRows) ;
          }
          col2im_indexed_dispatch(gpuMode,
                                  derData.memory + derDataOffset,
                                  temp.memory,
                                  convIndices.memoryInt,
                                  convIndices.geom.numElements,
                                  data.geom.height, data.geom.width, data.geom.depth, numImages,
                                  filters.geom.height, filters.geom.width);
        }
      } else {
        float *curOutputMemory = numImages > 1 ? outputMasked.memory : output.memory + outputOffset;

        im2col_indexed_dispatch(gpuMode,
                                temp.memory,
                                data.memory + dataOffset,
                                convIndices.memoryInt,
                                convIndices.geom.numElements,
                                data.geom.height, data.geom.width, data.geom.depth, numImages,
                                filters.geom.height, filters.geom.width) ;
        for (int g = 0 ; g < numGroups ; ++ g) {
          ptrdiff_t filterGrpOffset = k * n * g ;
          ptrdiff_t tempGrpOffset = numRows * k * g ;
          ptrdiff_t outputGrpOffset = numRows * n * g  ;
          float alpha = 1 ;
          float beta = 0 ;
          sgemm_dispatch(gpuMode, 'n', 'n',
                         numRows, n, k,
                         alpha,
                         temp.memory + tempGrpOffset, numRows,
                         filters.memory + filterGrpOffset, k,
                         beta,
                         curOutputMemory + outputGrpOffset,
                         numRows) ;
        }
        if (hasBiases) {
          float alpha = 1 ;
          float beta = 1 ;
          ptrdiff_t q = 1 ;
          sgemm_dispatch(gpuMode, 'n', 'n',
                         numRows, biases.geom.numElements, q,
                         alpha,
                         allOnes.memory, numRows,
                         biases.memory, q,
                         beta,
                         curOutputMemory,
                         numRows) ;
        }

        if (numImages > 1) {
          transpose23_dispatch(gpuMode,
                               output.memory + outputOffset,
                               outputMasked.memory,
                               m, numImages, output.geom.depth) ;
        }
      }
    }
  } else {
    // This branch catches corner cases: 1x1 convolutions (skipping im2col/col2im), and when
    // vl_nnconv called without convIndices.
    // It can be merged with the previous branch, but the number of conditionals inside is already
    // way too high.
    for (int image = 0 ; image < data.geom.size ; ++image) {
      /*
       temp (phi(x)): m x k
       filters, derFilters: k x n (for one group of filters)
       derOutput (dzdy) : m x n (for one group of filters)
       res (y) : m x n (for one group of filters)
       */
      ptrdiff_t dataOffset = (data.geom.height*data.geom.width*data.geom.depth) * image ;
      ptrdiff_t outputOffset = (output.geom.height*output.geom.width*output.geom.depth) * image ;
      ptrdiff_t derDataOffset = (derData.geom.height*derData.geom.width*derData.geom.depth) * image ;
      ptrdiff_t derOutputOffset = (derOutput.geom.height*derOutput.geom.width*derOutput.geom.depth) * image ;
      ptrdiff_t m = outputGeom.height * outputGeom.width ; /* num output pixels */
      ptrdiff_t n = filters.geom.size/numGroups ; /* num filters per group */
      ptrdiff_t k = filters.geom.height*filters.geom.width*filters.geom.depth ; /* filter volume */

      float* tempMemory;

      if (backMode) {
        /* ---------------------------------------------------------- */
        /*                                              Backward mode */
        /* ---------------------------------------------------------- */

        /* compute derFilters dz/dF */
        if (computeDerFilters) {
          if (!is_1x1) {
            im2col_dispatch(gpuMode,
                            temp.memory,
                            data.memory + dataOffset,
                            data.geom.height, data.geom.width, data.geom.depth,
                            filters.geom.height, filters.geom.width,
                            strideY, strideX,
                            padTop, padBottom, padLeft, padRight) ;
            tempMemory = temp.memory;
          } else {
            tempMemory = data.memory + dataOffset;
          }
          for (int g = 0 ; g < numGroups ; ++ g) {
            ptrdiff_t filterGrpOffset = k * n * g ;
            ptrdiff_t tempGrpOffset = m * k * g ;
            ptrdiff_t derOutputGrpOffset = m * n * g ;
            float alpha = 1 ;
            float beta = (image > 0 || derFiltersInitialized) ; /* this saves init. the output array with 0 */
            sgemm_dispatch(gpuMode, 't', 'n',
                           k, n, m,
                           alpha,
                           tempMemory + tempGrpOffset, m,
                           derOutput.memory + derOutputOffset + derOutputGrpOffset, m,
                           beta,
                           derFilters.memory + filterGrpOffset, k) ;
          }
        }

        /* compute derData dz/dbias */
        if (computeDerBiases & hasBiases) {
          sgemv_dispatch(gpuMode, 't',
                         m, filters.geom.size,
                         1, /* alpha */
                         derOutput.memory + derOutputOffset, m,
                         allOnes.memory, 1,
                         (float)(image > 0 || derBiasesInitialized), /* beta */
                         derBiases.memory, 1) ;
        }

        /* compute derData dz/dx */
        if (computeDerData) {
          if (!is_1x1) {
            tempMemory = temp.memory;
          } else {
            tempMemory = derData.memory + derDataOffset;
          }

          for (int g = 0 ; g < numGroups ; ++ g) {
            ptrdiff_t filterGrpOffset = k * n * g ;
            ptrdiff_t tempGrpOffset = m * k * g ;
            ptrdiff_t derOutputGrpOffset = m * n * g ;
            float alpha = 1 ;
            float beta = 0 ;
            sgemm_dispatch(gpuMode, 'n', 't',
                           m, k, n,
                           alpha,
                           derOutput.memory + derOutputOffset + derOutputGrpOffset, m,
                           filters.memory + filterGrpOffset, k,
                           beta,
                           tempMemory + tempGrpOffset,
                           m) ;
          }
          if (!is_1x1) {
            col2im_dispatch(gpuMode,
                            derData.memory + derDataOffset,
                            temp.memory,
                            data.geom.height, data.geom.width, data.geom.depth,
                            filters.geom.height, filters.geom.width,
                            strideY, strideX,
                            padTop, padBottom, padLeft, padRight) ;
          }
        }
      } else {
        /* ---------------------------------------------------------- */
        /*                                               Forward mode */
        /* ---------------------------------------------------------- */
        if (!is_1x1) {
          im2col_dispatch(gpuMode,
                          temp.memory,
                          data.memory + dataOffset,
                          data.geom.height, data.geom.width, data.geom.depth,
                          filters.geom.height, filters.geom.width,
                          strideY, strideX,
                          padTop, padBottom, padLeft, padRight) ;
          tempMemory = temp.memory;
        } else {
          tempMemory = data.memory + dataOffset;
        }
        for (int g = 0 ; g < numGroups ; ++ g) {
          ptrdiff_t filterGrpOffset = k * n * g ;
          ptrdiff_t tempGrpOffset = m * k * g ;
          ptrdiff_t outputGrpOffset = m * n * g  ;
          float alpha = 1 ;
          float beta = 0 ;
          sgemm_dispatch(gpuMode, 'n', 'n',
                         m, n, k,
                         alpha,
                         tempMemory + tempGrpOffset, m,
                         filters.memory + filterGrpOffset, k,
                         beta,
                         output.memory + outputOffset + outputGrpOffset, m) ;
        }
        if (hasBiases) {
          float alpha = 1 ;
          float beta = 1 ;
          ptrdiff_t q = 1 ;
          sgemm_dispatch(gpuMode, 'n', 'n',
                         m, biases.geom.numElements, q,
                         alpha,
                         allOnes.memory, m,
                         biases.memory, q,
                         beta,
                         output.memory + outputOffset, m) ;
        }
      }
    }
  }

  /* -------------------------------------------------------------- */
  /*                                                        Cleanup */
  /* -------------------------------------------------------------- */

  packed_data_deinit(&data) ;
  packed_data_deinit(&filters) ;
  packed_data_deinit(&biases) ;
  if (convIndicesMode) {
    packed_data_deinit(&convIndices);
  }
  if (backMode) {
    packed_data_deinit(&derOutput) ;
    out[OUT_RESULT] = (computeDerData) ? packed_data_deinit_extracting_array(&derData) : mxCreateDoubleMatrix(0,0,mxREAL) ;
    out[OUT_DERFILTERS] =(computeDerFilters)? packed_data_deinit_extracting_array(&derFilters) : mxCreateDoubleMatrix(0,0,mxREAL) ;
    out[OUT_DERBIASES] = (computeDerBiases & hasBiases) ? packed_data_deinit_extracting_array(&derBiases) : mxCreateDoubleMatrix(0,0,mxREAL) ;
  } else {
    out[OUT_RESULT] = packed_data_deinit_extracting_array(&output) ;
  }
  packed_data_deinit(&derFiltersInit) ;
  packed_data_deinit(&derBiasesInit) ;
}
