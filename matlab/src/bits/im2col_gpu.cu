#include "hip/hip_runtime.h"
/** @file im2col.cu
 ** @brief Image to columns and back (GPU)
 ** @author Andrea Vedaldi
 **/

/*
Copyright (C) 2014 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "im2col.hpp"
#include "gpu.hpp"

/* ---------------------------------------------------------------- */
/*                                                     im2col (GPU) */
/* ---------------------------------------------------------------- */

template <typename T>
__global__ void
im2col_gpu_kernel(T* stacked,
                  T const* data,
                  const int numPatchesX,
                  const int numPatchesY,
                  const int numPatchSlices,
                  const int width,
                  const int height,
                  const int windowWidth,
                  const int windowHeight,
                  const int strideX,
                  const int strideY,
                  const int padLeft,
                  const int padTop)
{
  /* each kernel copies the pixels in an image patch for one channel */
  int index = threadIdx.x + blockIdx.x * blockDim.x ;
  if (index < numPatchSlices) {
    /* 
      get the patch slice (x,y,z) to copy
     */
    int x = index ;
    int y = x / numPatchesX ;
    int z = y / numPatchesY ;
    x %= numPatchesX ;
    y %= numPatchesY ;

    /* 
     pick the top-left corer of the patch slice in the input image
     */
    int x_data = x * strideX - padLeft ;
    int y_data = y * strideY - padTop ;
    data += (z * height + y_data) * width + x_data ;

    /* 
     pick the column of the stacked image which contains this patch,
     and move down along the column at the beginning of the patch slice
     */
    int patchSliceOffset = (windowWidth*windowHeight) * z ;
    stacked += (numPatchesY * patchSliceOffset + y) * numPatchesX + x ;

    /*
     copy the patch slice
     */
    for (int v = 0 ; v < windowHeight ; ++v) {
      for (int u = 0 ; u < windowWidth ; ++u) {
        if (y_data + v >= 0 &&
            y_data + v < height &&
            x_data + u >= 0 &&
            x_data + u < width) {
          *stacked = data[v * width + u] ;
        } else {
          *stacked = 0 ;
        }
        stacked += (numPatchesX*numPatchesY) ;
      }
    }
  }
}

template <typename T>
void im2col_gpu(T* stacked,
                T const* data,
                size_t width,
                size_t height,
                size_t depth,
                size_t windowWidth,
                size_t windowHeight,
                size_t strideX,
                size_t strideY,
                size_t padLeft,
                size_t padRight,
                size_t padTop,
                size_t padBottom)
{
  int numPatchesX = (width + (padLeft + padRight) - windowWidth)/strideX + 1 ;
  int numPatchesY = (height + (padTop + padBottom) - windowHeight)/strideY + 1 ;
  int numPatchSlices = numPatchesX * numPatchesY * depth ;

  /*
   Each kernel copies a feature dimension of a patch.
   */
  im2col_gpu_kernel<T>
  <<< divideUpwards(numPatchSlices, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (stacked,
   data,
   numPatchesX,
   numPatchesY,
   numPatchSlices,
   width, height,
   windowWidth, windowHeight,
   strideX, strideY,
   padLeft, padTop) ;

  if (hipPeekAtLastError() != hipSuccess) {
    std::cout
    <<"im2col: CUDA kernel error ("
    <<hipGetErrorString(hipPeekAtLastError())
    <<")"<<std::endl ;
  }
}

// Explicit instantiation
template void im2col_gpu<float>(float* stacked,
                                float const* data,
                                size_t width,
                                size_t height,
                                size_t depth,
                                size_t windowWidth,
                                size_t windowHeight,
                                size_t strideX,
                                size_t strideY,
                                size_t padLeft,
                                size_t padRight,
                                size_t padTop,
                                size_t padBottom);

template void im2col_gpu<double>(double* stacked,
                                 double const* data,
                                 size_t width,
                                 size_t height,
                                 size_t depth,
                                 size_t windowWidth,
                                 size_t windowHeight,
                                 size_t strideX,
                                 size_t strideY,
                                 size_t padLeft,
                                 size_t padRight,
                                 size_t padTop,
                                 size_t padBottom);

/* ---------------------------------------------------------------- */
/*                                        im2col with indices (GPU) */
/* ---------------------------------------------------------------- */

template <typename T>
__global__ void
im2col_gpu_indexed_size_1_kernel(T* __restrict__ stacked,
                                 T const* __restrict__ data,
                                 int const* __restrict__ indices,
                                 const int indicesLength,
                                 const int numPatchSlices,
                                 const int dataSize)
{
  /* each kernel copies the pixels in an image patch for one channel */
  int index = threadIdx.x + blockIdx.x * blockDim.x ;
  if (index < numPatchSlices) {
    int x = index;
    int z = index / indicesLength;
    x %= indicesLength;

    int idxValue = indices[x];
    stacked[index] = (idxValue != -1) ? data[z * dataSize + idxValue] : 0;
  }
}

template <typename T>
__global__ void
im2col_gpu_indexed_kernel(T* __restrict__ stacked,
                          T const* __restrict__ data,
                          int const* __restrict__ indices,
                          const int maskIndicesLength,
                          const int dataSize,
                          const int depth,
                          const int depthCol,
                          const int size,
                          const int numPatchSlices)
{
  /* each kernel copies the pixels in an image patch for one channel */
  int index = threadIdx.x + blockIdx.x * blockDim.x ;
  if (index < numPatchSlices) {
    int x = index;
    int s = x / maskIndicesLength;
    int d = s / size;
    int c = d / depthCol;
    x %= maskIndicesLength;
    s %= size;
    d %= depthCol;

    int idxValue = indices[d * maskIndicesLength + x];
    stacked[index] = (idxValue != -1) ? data[(s * depth + c) * dataSize + idxValue] : 0;
  }
}

template <typename T>
void im2col_indexed_gpu(T* stacked,
                        T const* data,
                        int const* indices,
                        int indicesLength,
                        size_t width,
                        size_t height,
                        size_t depth,
                        size_t size,
                        size_t windowWidth,
                        size_t windowHeight)
{
  int numPatchSlices = indicesLength * depth * size ;
  int depthCol = windowWidth * windowHeight;
  int maskIndicesLength = indicesLength / depthCol;

  if (size == 1) {
    im2col_gpu_indexed_size_1_kernel<T>
    <<< divideUpwards(numPatchSlices, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
    (stacked,
     data,
     indices, indicesLength,
     numPatchSlices,
     width * height) ; 
  } else {
    im2col_gpu_indexed_kernel<T>
    <<< divideUpwards(numPatchSlices, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
    (stacked,
     data,
     indices,
     maskIndicesLength,
     width * height,
     depth,
     depthCol,
     size,
     numPatchSlices) ;
  }

  if (hipPeekAtLastError() != hipSuccess) {
    std::cout
    <<"im2col_indexed_gpu: CUDA kernel error ("
    <<hipGetErrorString(hipPeekAtLastError())
    <<")"<<std::endl ;
  }
}

// Explicit instantiation
template void im2col_indexed_gpu<float>(float* stacked,
                                        float const* data,
                                        int const* indices,
                                        int indicesLength,
                                        size_t width,
                                        size_t height,
                                        size_t depth,
                                        size_t size,
                                        size_t windowWidth,
                                        size_t windowHeight);

template void im2col_indexed_gpu<double>(double* stacked,
                                         double const* data,
                                         int const* indices,
                                         int indicesLength,
                                         size_t width,
                                         size_t height,
                                         size_t depth,
                                         size_t size,
                                         size_t windowWidth,
                                         size_t windowHeight);

/* ---------------------------------------------------------------- */
/*                                                     col2im (GPU) */
/* ---------------------------------------------------------------- */

template <typename T>
__global__ void col2im_gpu_kernel(T* data,
                                  T const* stacked,
                                  const int numPatchesX,
                                  const int numPatchesY,
                                  const int dataVolume,
                                  const int width,
                                  const int height,
                                  const int depth,
                                  const int windowWidth,
                                  const int windowHeight,
                                  const int strideX,
                                  const int strideY,
                                  const int padLeft,
                                  const int padTop)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < dataVolume)
  {
    T accumulator = 0 ;
    /*
     This kernel accumulates on data[index] all elements in stacked
     that receive copies of data[index] in im2col.
     
     Consider coordinate (x_data,y_data) in the input image. Relative to patch
     (x,y), this has offset
     
     u = x_data - (x * strideX - padLeft)
     v = y_data - (y * strideY - padRight)
     
     In particular, (x_data,y_data) is contained (and hence contributes)
     to patch (x,y) if, and only if,
     
     0 <= u < windowWidth  <==>  1) x_data >= x * strideX - padLeft
                                 2) x_data <  x * strideX - padLeft + windowWidth
     
     and similar for y.
     
     Hence, the patches that contribute to (x_data,y_data) are given
     by indexes (x,y) such that
     
     (x_data + padLeft - windowWidth)/stride < x
         <= (x_data + padLeft)/stride
     
     or, accounting for the boundaries,

       x1 <= x <= x2, such that
         x1 = max(0,  1 + floor(x_data + padLeft - windowWidth)/stride),
         x2 = min(numPatchesX-1,  floor(x_data + padLeft)/stride),
     
     and similar for y.
     
     Note that (x_data + padLeft - windowWidth) may be negative. In this case,
     the C convention for rounding division towards zero fails to compute
     the floor() properly. Instead, we check this case explicitly and set
     */

    int x_data = index ;
    int y_data = x_data / width ;
    int z = y_data / height ;
    x_data %= width ;
    y_data %= height ;

    int dx = x_data + padLeft - windowWidth ;
    int dy = y_data + padTop - windowHeight ;
    int x1 = (dx >= 0) ? dx/strideX + 1 : 0 ;
    int y1 = (dy >= 0) ? dy/strideY + 1 : 0 ;
    int x2 = min((x_data + padLeft) / strideX, numPatchesX - 1) ;
    int y2 = min((y_data + padTop) / strideY, numPatchesY - 1) ;

    /*
     Knowing which patches (x,y) contribute to (x_data,y_data) is not enough;
     we need to determine the specific element within each patch. This
     is given by the offset as given above:
     
     u(x) = x_data - (x * strideX - padLeft)
     v(y) = y_data - (y * strideY - padRight)
     
     Now we can comptute the indeces of the elements of stacked[] to accumulate:
     
     stackedIndex(x,y) = 
         (y * numPatchesX + x) +                 // column offset
         ((z * windowHeight + v(y)) * windowWidth + u(x)) *  // within patch offset
            (numPatchesX*numPatchesY)

     Substituting the expression fo u(x), we find

     stackedIndex(x,y) =
         = (y * numPatchesX + x)
         + ((z * windowHeight + y_data + padTop) * windowWidth + x_data + padLeft)
           * (numPatchesX*numPatchesY)
         - ((y * strideY) * windowWidth + x * strideX)
           * (numPatchesX*numPatchesY)
         = (z * windowHeight + y_data + padTop) * windowWidth + x_data + padLeft)
         + x * (1 - strideX*numPatchesY*numPatchesX)
         + y * (1 - strideY*numPatchesY*windowWidth)*numPatchesX ;

     */

    int deltax = (1 - strideX * numPatchesY * numPatchesX) ;
    int deltay = (1 - strideY * numPatchesY * windowWidth) * numPatchesX ;
    stacked += ((z * windowHeight + y_data + padTop) * windowWidth + (x_data + padLeft)) * (numPatchesX*numPatchesY) ;

    for (int y = y1 ; y <= y2 ; ++ y) {
      for (int x = x1 ; x <= x2 ; ++ x) {
        accumulator += stacked[y * deltay + x * deltax];
      }
    }
    data[index] = accumulator;
  }
}

template <typename T>
void col2im_gpu(T* data,
                T const* stacked,
                size_t width,
                size_t height,
                size_t depth,
                size_t windowWidth,
                size_t windowHeight,
                size_t strideX,
                size_t strideY,
                size_t padLeft,
                size_t padRight,
                size_t padTop,
                size_t padBottom)
{
  /*
   each kernel integrates all contributions to a particular element
   of data.
   */
  int numPatchesX = (width + (padLeft + padRight) - windowWidth)/strideX + 1 ;
  int numPatchesY = (height + (padTop + padBottom) - windowHeight)/strideY + 1 ;
  int dataVolume = width * height * depth ;

  col2im_gpu_kernel<T>
  <<< divideUpwards(dataVolume, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (data,
   stacked,
   numPatchesX,
   numPatchesY,
   dataVolume,
   width, height, depth,
   windowWidth, windowHeight,
   strideX, strideY,
   padLeft, padTop) ;

  if (hipPeekAtLastError() != hipSuccess) {
    std::cout
    <<"col2im: CUDA kernel error ("
    <<hipGetErrorString(hipPeekAtLastError())
    <<")"<<std::endl ;
  }
}

template void col2im_gpu<float>(float* data,
                                float const* stacked,
                                size_t width,
                                size_t height,
                                size_t depth,
                                size_t windowWidth,
                                size_t windowHeight,
                                size_t strideX,
                                size_t strideY,
                                size_t padLeft,
                                size_t padRight,
                                size_t padTop,
                                size_t padBottom);

template void col2im_gpu<double>(double* data,
                                 double const* stacked,
                                 size_t width,
                                 size_t height,
                                 size_t depth,
                                 size_t windowWidth,
                                 size_t windowHeight,
                                 size_t strideX,
                                 size_t strideY,
                                 size_t padLeft,
                                 size_t padRight,
                                 size_t padTop,
                                 size_t padBottom);

/* ---------------------------------------------------------------- */
/*                                        col2im with indices (GPU) */
/* ---------------------------------------------------------------- */

template <typename T>
__global__ void
col2im_gpu_indexed_size_1_kernel(T* __restrict__ data,
                                 T const* __restrict__ stacked,
                                 int const* __restrict__ indices,
                                 const int indicesLength,
                                 const int numPatchSlices,
                                 const int width,
                                 const int height,
                                 const int depth)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x ;
  if (index < numPatchSlices) {
    int idx = index % indicesLength;
    int z = index / indicesLength;

    int idxValue = indices[idx];
    if (idxValue != -1) {
      atomicAdd(data + z * width * height + idxValue, stacked[index]);
    }
  }
}

template <typename T>
__global__ void
col2im_gpu_indexed_kernel(T* __restrict__ data,
                          T const* __restrict__ stacked,
                          int const* __restrict__ indices,
                          const int maskIndicesLength,
                          const int dataSize,
                          const int depth,
                          const int depthCol,
                          const int size,
                          const int numPatchSlices)
{
  /* each kernel copies the pixels in an image patch for one channel */
  int index = threadIdx.x + blockIdx.x * blockDim.x ;
  if (index < numPatchSlices) {
    int x = index;
    int s = x / maskIndicesLength;
    int d = s / size;
    int c = d / depthCol;
    x %= maskIndicesLength;
    s %= size;
    d %= depthCol;

    int idxValue = indices[d * maskIndicesLength + x];
    if (idxValue != -1) {
      atomicAdd(data + (s * depth + c) * dataSize + idxValue, stacked[index]) ;
    }
  }
}

template <typename T>
void col2im_indexed_gpu(T* data,
                        T const* stacked,
                        int const* indices,
                        int indicesLength,
                        size_t width,
                        size_t height,
                        size_t depth,
                        size_t size,
                        size_t windowWidth,
                        size_t windowHeight)
{
  int numPatchSlices = indicesLength * depth * size ;
  int depthCol = windowWidth * windowHeight;
  int maskIndicesLength = indicesLength / depthCol;

  hipMemset(data, 0, sizeof(T)*width*height*depth*size);

  if (size == 1) {
    col2im_gpu_indexed_size_1_kernel<T>
    <<< divideUpwards(numPatchSlices, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
    (data,
     stacked,
     indices, indicesLength,
     numPatchSlices,
     width, height, depth) ;
  } else {
    col2im_gpu_indexed_kernel<T>
    <<< divideUpwards(numPatchSlices, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
    (data,
     stacked,
     indices,
     maskIndicesLength,
     width * height,
     depth,
     depthCol,
     size,
     numPatchSlices) ;
  }

  if (hipPeekAtLastError() != hipSuccess) {
    std::cout
    <<"col2im_indexed_gpu: CUDA kernel error ("
    <<hipGetErrorString(hipPeekAtLastError())
    <<")"<<std::endl ;
  }
}

// Explicit instantiation
template void col2im_indexed_gpu<float>(float* data,
                                        float const* stacked,
                                        int const* indices,
                                        int indicesLength,
                                        size_t width,
                                        size_t height,
                                        size_t depth,
                                        size_t size,
                                        size_t windowWidth,
                                        size_t windowHeight);

template <typename T>
__global__ void transpose23_kernel(T* transposed,
                                   const T* data,
                                   const int d1,
                                   const int d2,
                                   const int d3,
                                   const int numPatchSlices)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x ;
  if (index < numPatchSlices) {
    int x = index;
    int y = x / d1;
    int z = y / d2;
    x %= d1;
    y %= d2;

    transposed[y*(d1*d3) + z*d1 + x] = data[z*(d1*d2) + y*d1 + x];
  }
}

template <typename T>
void transpose23_gpu(T* transposed,
                     T const* data,
                     size_t d1,
                     size_t d2,
                     size_t d3)
{
  int numPatchSlices = d1 * d2 * d3 ;

  /*
   Each kernel copies a feature dimension of a patch.
   */
  transpose23_kernel<T>
  <<< divideUpwards(numPatchSlices, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (transposed,
   data,
   d1,
   d2,
   d3,
   numPatchSlices) ;

  if (hipPeekAtLastError() != hipSuccess) {
    std::cout
    <<"transpose23_gpu: CUDA kernel error ("
    <<hipGetErrorString(hipPeekAtLastError())
    <<")"<<std::endl ;
  }
}

template void transpose23_gpu<float>(float* transposed,
                                     float const* data,
                                     size_t d1,
                                     size_t d2,
                                     size_t d3);

template void transpose23_gpu<double>(double* transposed,
                                      double const* data,
                                      size_t d1,
                                      size_t d2,
                                      size_t d3);
